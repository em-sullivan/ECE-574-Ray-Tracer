#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <chrono>
#include <cstdlib>
#include <hiprand/hiprand_kernel.h>
#include "Vec3.h"
#include "Color.h"
#include "Ray.h"
#include "Sphere.h"
#include "Hittable_List.h"
#include "Camera.h"
#include "Texture.h"
#include "Render.h"
#include "Moving-Sphere.h"

#include "shader_stb_image.h"

using namespace std::chrono;

#define RND (hiprand_uniform(&local_rand_state))

__global__ void glow_balls(Hittable **d_list, Hittable **d_world, Camera **d_camera, int nx, int ny, hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        int i = 0;
        d_list[i++] = new Sphere(Vec3(0, -1000, -1), 1000, new Lambertian(new Solid_Color(0.5, 0.5, 0.5)));
        
        for (int a = -8; a < 8; a++) {
            for (int b = -8; b < 8; b++) {
                float choose_mat = RND;
                Vec3 center(a + 0.9f * RND, 0.2, b + 0.9f * RND);
                if (choose_mat < .5f) {
                    d_list[i++] = new Sphere(center, 0.2, 
                        new Diffuse_Light(new Solid_Color(RND * RND, 0, RND * RND)));
                } else {
                    d_list[i++] = new Moving_Sphere(center, center + Vec3(0, 0, RND), 0.0, 1.0, 0.2,
                        new Diffuse_Light(new Solid_Color(RND * RND, 0, RND * RND)));
                }

            }
        }

        d_list[i++] = new Sphere(Vec3(4, 1, 1.5f), 1.0, new Dielectric(1.5));

        // World
        *rand_state = local_rand_state;
        *d_world = new Hittable_List(d_list, 16 * 16 + 2);

        // Camera
        Vec3 lookfrom = Vec3(13, 2, 5);
        Vec3 lookat = Vec3(0, 0, 0);
        float dist_to_focus = 10.0;
        float aperture = .1f;
        *d_camera = new Camera(lookfrom, lookat, Vec3(0,1,0), 25.0, float(nx)/float(ny), aperture, dist_to_focus, 0 ,1);
    }
}

__global__ void free_glow_balls(Hittable **d_list, Hittable **d_world, Camera **d_camera) 
{
    for(int i=0; i < (16 * 16 + 2); i++) {
        delete ((Hittable *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__global__ void create_world(Hittable **d_list, Hittable **d_world, Camera **d_camera, int nx, int ny, hiprandState *rand_state) 
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new Sphere(Vec3(0,-1000.0,-1), 1000, new Lambertian(new Solid_Color(Vec3(0.5,0.5,0.5))));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                Vec3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new Sphere(center, 0.2, new Lambertian(new Solid_Color(Vec3(RND*RND, RND*RND, RND*RND))));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new Sphere(center, 0.2, new Metal(Vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new Sphere(center, 0.2, new Dielectric(1.5));
                }
            }
        }

        d_list[i++] = new Sphere(Vec3(0, 1,0),  1.0, new Dielectric(1.5));
        d_list[i++] = new Sphere(Vec3(-4, 1, 0), 1.0 , new Lambertian(new Solid_Color(Vec3(0.4,0.3,0.1))));
        //d_list[i++] =  new Sphere(Vec3(0, 4, 5),  1.0, new Diffuse_Light( new Solid_Color(Vec3(7, 7, 7))));
        d_list[i++] = new Sphere(Vec3(4, 1, 0),  1.0, new Metal(Vec3(0.7, 0.6, 0.5), 0.0));

        
        *rand_state = local_rand_state;
        *d_world  = new Hittable_List(d_list, 22*22+1+3);

        Vec3 lookfrom = Vec3(13,2,3);
        Vec3 lookat = Vec3(0,0,0);
        float dist_to_focus = 10.0;
        float aperture = 0.1;
        *d_camera   = new Camera(lookfrom, lookat, Vec3(0,1,0), 25.0, float(nx)/float(ny), aperture, dist_to_focus, 0 ,1);
    }
}

__global__ void free_world(Hittable **d_list, Hittable **d_world, Camera **d_camera) 
{
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((Sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__global__ void solar_system(Hittable **d_list, Hittable **d_world, Camera **d_camera, int nx, int ny, hiprandState *rand_state, int tex_nx, int tex_ny, int texHQ_nx, int texHQ_ny, unsigned char *sun,  
                                              unsigned char *mercury, unsigned char *venus, unsigned char *earth,  unsigned char *mars,  unsigned char *jupiter,  unsigned char *saturn,  unsigned char *uranus,  unsigned char *neptune, unsigned char* pluto) 
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        int i = 0;
        Texture *sunText = new Image_Text(sun, texHQ_nx, texHQ_ny);
        Texture *star1Text = new Solid_Color(Vec3(1, 1, 1));                // White 
        Texture *star2Text = new Solid_Color(Vec3(0.75, 0.6, 0.5));     // Yellow
        Texture *star3Text = new Solid_Color(Vec3(0.93, 0.41, 0.24)); // Red
        Texture *star4Text = new Solid_Color(Vec3(0.4, .82, 0.95));    // Blue 

        // Create sun and slightly bigger light source
        d_list[i++] = new Sphere(Vec3(0, 0, -320), 300.0 , new Diffuse_Light(sunText));
        d_list[i++] = new Sphere(Vec3(0, 0, -1300), 600.0 , new Diffuse_Light(new Solid_Color(Vec3(0.25, 0.2, 0.12))));

        // Create each planet in a line
        d_list[i++] = new Sphere(Vec3(0, 0, -10), 2, new Lambertian(new Image_Text(mercury, tex_nx, tex_ny)));
        d_list[i++] = new Sphere(Vec3(0, 0, 0), 3.6, new  Lambertian(new Image_Text(venus, tex_nx, tex_ny)));
        d_list[i++] = new Sphere(Vec3(0, 0, 13), 4.4, new  Lambertian(new Image_Text(earth, tex_nx, tex_ny)));
        d_list[i++] = new Sphere(Vec3(0, 0, 27), 2.4, new  Lambertian(new Image_Text(mars, tex_nx, tex_ny)));
        d_list[i++] = new Sphere(Vec3(0, 0, 80), 34.0, new  Lambertian(new Image_Text(jupiter, texHQ_nx, texHQ_ny)));
        d_list[i++] = new Sphere(Vec3(0, 0, 190), 28.0, new  Lambertian(new Image_Text(saturn, texHQ_nx, texHQ_ny)));
        d_list[i++] = new Sphere(Vec3(0, 0, 310), 16.4 , new  Lambertian(new Image_Text(uranus, tex_nx, tex_ny)));
        d_list[i++] = new Sphere(Vec3(0, 0, 450),  16.0, new  Lambertian(new Image_Text(neptune, tex_nx, tex_ny)));
        d_list[i++] = new Sphere(Vec3(0, 0, 575),  2.75, new  Lambertian(new Image_Text(pluto, tex_nx, tex_ny)));


        // Generates random stars in the background
        // DEPENDS GREATLY on lookfrom, lookat, and fov
        for(int a = -450; a < 450; a+=20) {
                for(int c = -20; c < 1100; c+=20) { 
                    float starColor = RND;
                    
                    float rand1 = RND;
                    rand1 *= (20.f+0.999999f);
                    rand1 = truncf(rand1);
                    
                    float rand2 = RND;
                    rand2 *= (20.f+0.999999f);
                    rand2 = truncf(rand2);

                    float rand3 = RND;
                    rand3 *= (20.f+0.999999f);
                    rand3 = truncf(rand3);
                    
                    Vec3 center(250 + rand1 + (800 - c), a+rand2,  c+rand3);
                    if (starColor < 0.7f) {
                        d_list[i++] = new Sphere(center, RND, new Diffuse_Light(star1Text));
                    } else if  (starColor < 0.9f) {
                        d_list[i++] = new Sphere(center, RND, new Diffuse_Light(star2Text));
                    } else if  (starColor < 0.95f) {
                        d_list[i++] = new Sphere(center, RND, new Diffuse_Light(star3Text));
                    } else {
                        d_list[i++] = new Sphere(center, RND, new Diffuse_Light(star4Text));
                    }
                }
            }

        *rand_state = local_rand_state;
        *d_world  = new Hittable_List(d_list, 11+45*56);

        Vec3 lookfrom = Vec3(-145,0, -25);
        Vec3 lookat = Vec3(-110,0, 5);
        float dist_to_focus = 100.0;
        float aperture = 0.1;
        float fov = 52.0;
        *d_camera   = new Camera(lookfrom, lookat, Vec3(0,1,0), fov, float(nx)/float(ny), aperture, dist_to_focus, 0 ,1);
    }
}

__global__ void free_solar_system(Hittable **d_list, Hittable **d_world, Camera **d_camera) 
{
    for(int i=0; i < 11+45*56; i++) {
        delete ((Hittable *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__global__ void pool_table(Hittable **d_list, Hittable **d_world, Camera **d_camera, int nx, int ny, hiprandState *rand_state) 
{

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;

        int i = 0;

        // Pool balls
        d_list[i++] = new Sphere(Vec3(-.05, 0, 2), .5,  new Metal(Vec3(4*0.8314, 4*0.83,  4*0.09), 1));
        d_list[i++] = new Sphere(Vec3(-.1, .5, 2.85), .5, new Metal(Vec3(4*0.059, 4*0.333, 4*0.694), 1));
        d_list[i++] = new Sphere(Vec3(-.1, -.5, 2.85), .5, new Metal(Vec3(4*0.73, 4*0.102, 4*0.102), 1));
        d_list[i++] = new Sphere(Vec3(-.15, -1, 3.7), .5, new Metal(Vec3(4*0.431, 4*0.102, 4*0.53), 1));
        d_list[i++] = new Sphere(Vec3(-.15, 0, 3.7), .5, new Metal(Vec3(0, 0, 0), 1));
        d_list[i++] = new Sphere(Vec3(-.15, 1, 3.7), .5, new Metal(Vec3(4*0.059, 4*0.302, 4*0.059), 1));
        d_list[i++] = new Sphere(Vec3(-.185, 1.5, 4.55), .5, new Metal(Vec3(4*0.37, 4*0.02,  4*0.01568), 1));
        d_list[i++] = new Sphere(Vec3(-.185, .5, 4.55), .5, new Metal(Vec3(4*0.80, 4*0.333, 4*0.063), 1));
        d_list[i++] = new Sphere(Vec3(-.185, -.5, 4.55), .5, new Metal(Vec3(4*0.8314, 4*0.83,  4*0.09), 1));
        d_list[i++] = new Sphere(Vec3(-.185, -1.5, 4.55), .5, new Metal(Vec3(4*0.059, 4*0.333, 4*0.694), 1));
        d_list[i++] = new Sphere(Vec3(-.25, -2.0, 5.40), .5, new Metal(Vec3(4*0.37, 4*0.02,  4*0.01568), 1));
        d_list[i++] = new Sphere(Vec3(-.25, -1.0, 5.40), .5, new Metal(Vec3(4*0.059, 4*0.302, 4*0.059), 1));
        d_list[i++] = new Sphere(Vec3(-.25, 0.0,  5.40), .5, new Metal(Vec3(4*0.80, 4*0.333, 4*0.063), 1));
        d_list[i++] = new Sphere(Vec3(-.25, 1.0, 5.40), .5, new Metal(Vec3(4*0.431, 4*0.102, 4*0.53), 1));
        d_list[i++] = new Sphere(Vec3(-.25, 2.0, 5.40), .5, new Metal(Vec3(4*0.73, 4*0.102, 4*0.102), 1));
        
        // Cue ball
        d_list[i++] = new Sphere(Vec3(0, 0, -5), .5, new Metal(Vec3(1, 1, 1), 1));

        // The ground of the table
        d_list[i++] = new Sphere(Vec3(-100.5, 0.0, -1.0), 100, new Lambertian(Vec3(.212, .4706, .294)));

        // Lighting above the table
        d_list[i++] = new Sphere(Vec3(100, 5, 0), 10, new Diffuse_Light(Vec3(20, 20, 20)));

        // Table
        *rand_state = local_rand_state;
        *d_world = new Hittable_List(d_list, 18);

        // Camera
        Vec3 lookfrom = Vec3(3, 0, -15);
        Vec3 lookat = Vec3(-2, 0, 10);
        float dist_to_focus = 25.0;
        float aperture = 0;
        *d_camera = new Camera(lookfrom, lookat, Vec3(0,0,1), 10.0, float(nx)/float(ny), aperture, dist_to_focus, 0 ,1);
    }
}

__global__ void free_pool_table(Hittable **d_list, Hittable **d_world, Camera **d_camera) 
{
    for(int i=0; i < 17; i++) {
        delete ((Hittable *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

int main(int argc, char **argv)
{
    auto program_start = high_resolution_clock::now();

    /****** Set up image size, block size, and frame buffer ******/
    int depth = 50;
    int tx = 8;
    int ty = 8;

    int image, nx, ny, ns;
    if (argc < 5) {
        nx = 400;
        ny = 225;
        ns = 10000;
        image = 2;    
    } else {
        image = atoi(argv[1]);
        nx = atoi(argv[2]);
        ny = atoi(argv[3]);
        ns = atoi(argv[4]);
    }
    

    /****** Allocate and copy memory for any image textures ******/
    int tex_nx, tex_ny, tex_nn;
    int texHQ_nx, texHQ_ny, texHQ_nn;
    unsigned char *dev_mercury;
    unsigned char *dev_venus;
    unsigned char *dev_earth;
    unsigned char *dev_mars;
    unsigned char *dev_jupiter;
    unsigned char *dev_saturn;
    unsigned char *dev_uranus;
    unsigned char *dev_neptune;
    unsigned char *dev_sun;
    unsigned char *dev_pluto;

    auto texture_time_start = high_resolution_clock::now();

    if (image == 1) {
        /******  Standard quality textures ******/
        unsigned char *mercury = stbi_load("../Common/textures/mercury.jpg", &tex_nx, &tex_ny, &tex_nn, 0);
        unsigned char *venus = stbi_load("../Common/textures/venus.jpg", &tex_nx, &tex_ny, &tex_nn, 0);
        unsigned char *earth = stbi_load("../Common/textures/earth.jpg", &tex_nx, &tex_ny, &tex_nn, 0);
        unsigned char *mars = stbi_load("../Common/textures/mars.jpg", &tex_nx, &tex_ny, &tex_nn, 0);
        unsigned char *uranus = stbi_load("../Common/textures/uranus.jpg", &tex_nx, &tex_ny, &tex_nn, 0);
        unsigned char *neptune = stbi_load("../Common/textures/neptune.jpg", &tex_nx, &tex_ny, &tex_nn, 0);
        unsigned char *pluto = stbi_load("../Common/textures/pluto.jpg", &tex_nx, &tex_ny, &tex_nn, 0);

        /****** High quality textures for larger bodies ******/
        unsigned char *sun = stbi_load("../Common/textures/sunHQ.jpg", &texHQ_nx, &texHQ_ny, &texHQ_nn, 0);
        unsigned char *jupiter = stbi_load("../Common/textures/jupiterHQ.jpg", &texHQ_nx, &texHQ_ny, &texHQ_nn, 0);
        unsigned char *saturn = stbi_load("../Common/textures/saturnHQ.jpg", &texHQ_nx, &texHQ_ny, &texHQ_nn, 0);


        /****** Allocate memory and copy each texture to the GPU ******/
        size_t texSize = tex_nx*tex_ny*tex_nn*sizeof(unsigned char);
        size_t texHQSize = texHQ_nx*texHQ_ny*texHQ_nn*sizeof(unsigned char);
    
        checkCudaErrors(hipMalloc((void **)&dev_mercury, texSize));
        checkCudaErrors(hipMemcpy(dev_mercury, mercury, texSize, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void **)&dev_venus, texSize));
        checkCudaErrors(hipMemcpy(dev_venus, venus, texSize, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void **)&dev_earth, texSize));
        checkCudaErrors(hipMemcpy(dev_earth, earth, texSize, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void **)&dev_mars, texSize));
        checkCudaErrors(hipMemcpy(dev_mars, mars, texSize, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void **)&dev_uranus, texSize));
        checkCudaErrors(hipMemcpy(dev_uranus, uranus, texSize, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void **)&dev_neptune, texSize));
        checkCudaErrors(hipMemcpy(dev_neptune, neptune, texSize, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void **)&dev_pluto, texSize));
        checkCudaErrors(hipMemcpy(dev_pluto, pluto, texSize, hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc((void **)&dev_sun, texHQSize));
        checkCudaErrors(hipMemcpy(dev_sun, sun, texHQSize, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void **)&dev_jupiter, texHQSize));
        checkCudaErrors(hipMemcpy(dev_jupiter, jupiter, texHQSize, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void **)&dev_saturn, texHQSize));
        checkCudaErrors(hipMemcpy(dev_saturn, saturn, texHQSize, hipMemcpyHostToDevice));
    }

    /****** Allocate and copy memory for adjustable background color ******/
    Color background;
    if (image == 0) background = Color(0, 0, 0);
    else if (image == 1) background = Color(0, 0, 0);
    else if (image == 2) background = Color(0, 0, 0);
    else background = Color(0.70, 0.80, 1.00);

    Color *dev_background;
    checkCudaErrors(hipMallocManaged((void **)&dev_background, sizeof(Color)));
    checkCudaErrors(hipMemcpy(dev_background, &background, sizeof(Color), hipMemcpyHostToDevice));

    auto texture_time_end = high_resolution_clock::now();


    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    auto create_time_start = high_resolution_clock::now();

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(Vec3);

    // allocate frame buffer (unified memory)
    Vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    /****** Render and time frame buffer ******/
    
    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hittables
    Hittable **d_list;
    int numHittables;

    if (image == 0) {
        numHittables = 16 * 16 + 2;
        checkCudaErrors(hipMalloc((void **)&d_list, numHittables*sizeof(Hittable *)));
    } else if (image == 1) {
        numHittables =  11+45*56;
        checkCudaErrors(hipMalloc((void **)&d_list, numHittables*sizeof(Hittable *))); 
    } else if (image == 2) {
        numHittables = 18;
        checkCudaErrors(hipMalloc((void **)&d_list, numHittables*sizeof(Hittable *))); 
    } else {
        numHittables = 22*22+1+4;
        checkCudaErrors(hipMalloc((void **)&d_list, numHittables*sizeof(Hittable *))); 
    }

    Hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Hittable *)));

    Camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));

    if (image == 0) {
        glow_balls<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    } else if (image == 1) {
        solar_system<<<1,1>>>(d_list,d_world,d_camera, nx, ny, d_rand_state2, tex_nx, tex_ny, texHQ_nx, texHQ_ny, dev_sun, 
                                                 dev_mercury, dev_venus, dev_earth, dev_mars, dev_jupiter, dev_saturn, dev_uranus, dev_neptune, dev_pluto);
    } else if (image == 2) {
        pool_table<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    } else {
        create_world<<<1,1>>>(d_list,d_world,d_camera, nx, ny, d_rand_state2);
    }

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto create_time_end = high_resolution_clock::now();
    std::cerr << "Starting Render.\n";

    auto render_time_start = high_resolution_clock::now();
    render<<<blocks, threads>>>(fb, nx, ny,  ns, d_camera, d_world, d_rand_state, depth, dev_background);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    
     std::cerr << "Render Finished.\n";

     auto render_time_end = high_resolution_clock::now();


    // Output File
    std::fstream file;
    file.open("out.ppm", std::ios::out);
   // std::streambuf *ppm_out = file.rdbuf();

     // Redirect Cout
   // std::cout.rdbuf(ppm_out);

    auto save_time_start = high_resolution_clock::now();

    // Output FB as Image
    file << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1;  j >= 0;  j--) {
        for (int i = 0;  i < nx;  i++) {
           size_t pixel_index = j*nx + i;
            writeColor(file,fb[pixel_index]);
        }
    }

    auto save_time_end = high_resolution_clock::now();

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    if (image == 0) {
        free_glow_balls<<<1,1>>>(d_list, d_world, d_camera);
    } else if (image == 1) {
        free_solar_system<<<1,1>>>(d_list, d_world, d_camera);
        checkCudaErrors(hipFree(dev_mercury));
        checkCudaErrors(hipFree(dev_venus));
        checkCudaErrors(hipFree(dev_earth));
        checkCudaErrors(hipFree(dev_mars));
        checkCudaErrors(hipFree(dev_jupiter));
        checkCudaErrors(hipFree(dev_saturn));
        checkCudaErrors(hipFree(dev_uranus));
        checkCudaErrors(hipFree(dev_neptune));
        checkCudaErrors(hipFree(dev_pluto));
        checkCudaErrors(hipFree(dev_sun));
    } else if (image == 2) {
        free_pool_table<<<1,1>>>(d_list, d_world, d_camera);
    }else {
        free_world<<<1,1>>>(d_list, d_world, d_camera);
    }

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(dev_background));

    std::cerr << "Image Successfully Saved." << std::endl;
    file.close();

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
    auto program_end = high_resolution_clock::now();

    // Texture Time 
    auto texture_time = duration_cast<milliseconds>(texture_time_end - texture_time_start);
    std::cout << "Texture Transfer Time: " << texture_time.count() << "ms" << std::endl;

    // Create Time 
    auto create_time = duration_cast<milliseconds>(create_time_end - create_time_start);
    std::cout << "World Creation Time: " << create_time.count() << "ms" << std::endl;

    // Render Time
    auto render_time = duration_cast<milliseconds>(render_time_end - render_time_start);
    std::cout << "Render Time: " << render_time.count() << "ms" << std::endl;

    // Save image time
    auto save_time = duration_cast<milliseconds>(save_time_end - save_time_start);
    std::cout << "Image Save Time: " << save_time.count() << "ms" << std::endl;
    
     // Total Time
    auto time = duration_cast<milliseconds>(program_end - program_start);
    std::cout << "Total Time: " << time.count() << "ms" << std::endl;
    return 0;
}